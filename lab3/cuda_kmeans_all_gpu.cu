#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"
#include <limits>
#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid() {
// return 0; /* TODO: copy me from naive */ /*  DONE */
    return blockIdx.x * blockDim.x + threadIdx.x;
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                               int numObjs,
                               int numClusters,
                               double *objects,     // [numCoords][numObjs]
                               double *clusters,    // [numCoords][numClusters]
                               int objectId,
                               int clusterId) {
  int i;
  double ans = 0.0;

  /* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */
  for (i = 0; i < numCoords; i++) {
    double diff = objects[objectId +  numObjs *  i] - clusters[clusterId +  numClusters * i];
    ans += diff * diff;
  }

  /* DONE */
  return (ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *deviceobjects,           //  [numCoords][numObjs]
/*
                          TODO: If you choose to do (some of) the new centroid calculation here, you will need some extra parameters here (from "update_centroids").
*/
                          double *deviceNewCentroids,    //  [numCoords][numClusters]
                          int *deviceNewSize,
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta) {
  extern __shared__ double shmemClusters[];

  /* TODO: copy me from shared version... */
  int  local_id = threadIdx.x;
  int  total = numClusters * numObjs ;
  if(local_id == 0) {
    // copy   clusters to shared memory  ;
        for(int  i = 0 ; i<total ; i++ ) {
            shmemClusters[i] = deviceClusters[i] ;
        }
      }
      //all threads should wait until  shared memory id ready
     __syncthreads();

      /* Get the global ID of the thread. */
      int tid = get_tid();

      /* TODO: copy me from shared version... */
      if (tid < numObjs) {
        int  index = 0;
        double dist =  euclid_dist_2_transpose( numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, index);
        double candDist ;
        for(int i=1 ; i<numClusters ;i++) {
            candDist = euclid_dist_2_transpose( numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, i);
            if(candDist < dist) {
                dist = candDist ;
                index = i ;
            }
        }
        if (deviceMembership[tid] != index) {
            atomicAdd(devdelta, 1.0);
        }
        /* assign the deviceMembership to object objectId */
        deviceMembership[tid] = index;

        /* TODO: copy me from shared version... */ /*  DONE */

        /* TODO: additional steps for calculating new centroids in GPU? */

        // Update  centroids
        for(int i=0 ;i<numCoords ; i++ ){
            atomicAdd(&(deviceNewCentroids[i * numClusters + index]) , deviceobjects[i*numObjs + index]) ;

        }
        // update  new size
        atomicAdd(&deviceNewSize[index] ,1) ;

      }
    }

    __global__ static
    void update_centroids(int numCoords,
                          int numClusters,
                          int *devicenewClusterSize,           //  [numClusters]
                          double *devicenewClusters,    //  [numCoords][numClusters]
                          double *deviceClusters)    //  [numCoords][numClusters])
    {

      /* TODO: additional steps for calculating new centroids in GPU? */

      /* Get the global ID of the thread. */
      int tid = get_tid();

      int coordId = tid % numCoords;// Which coordinate of the cluster
      int clusterId = tid / numCoords;// Which cluster this thread handles
      // remeber  centroid  is the mean point
      // so  the mean is the sum  which is  found and
      // updated  in  find_nearest_cluster / the  total
      // clusters -  the average point

      /*Boundary check */
      int check = clusterId < numClusters && coordId < numCoords ;
      if(check){
          int  clusterSize = devicenewClusterSize[clusterId] ;
          if(clusterSize > 0) {
              int  id  = coordId * numClusters + clusterId ;
              // find new  centroid
              deviceClusters[id] = devicenewClusters[id] / clusterSize ;
              // remeber  from find_nearest_cluster we have calculate
              // the  Sum of  cluster objects
              // line
              //atomicAdd(&(deviceNewCentroids[i * numClusters + index]) , deviceObjects[i*numObjs + index]) ;
              //so calculate  the  mean be dividd the cluster size
          }
      }
      // wait all to  finish
      __syncthreads();
      // update the sum of  centroids back to 0
      if(check){
        devicenewClusters[tid] = 0 ;
      }

    /*DONE*/

    }

    //
    //  ----------------------------------------
    //  DATA LAYOUT
    //
    //  objects         [numObjs][numCoords]
    //  clusters        [numClusters][numCoords]
    //  dimObjects      [numCoords][numObjs]
    //  dimClusters     [numCoords][numClusters]
    //  newClusters     [numCoords][numClusters]
    //  deviceObjects   [numCoords][numObjs]
    //  deviceClusters  [numCoords][numClusters]
    //  ----------------------------------------
    //
    /* return an array of cluster centers of size [numClusters][numCoords]       */
    void kmeans_gpu(double *objects,      /* in: [numObjs][numCoords] */
                    int numCoords,    /* no. features */
                    int numObjs,      /* no. objects */
                    int numClusters,  /* no. clusters */
                    double threshold,    /* % objects change membership */
                    long loop_threshold,   /* maximum number of iterations */
                    int *membership,   /* out: [numObjs] */
                    double *clusters,   /* out: [numClusters][numCoords] */
                    int blockSize) {
      double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0;
      double timing_gpu, timing_cpu, timing_transfers, transfers_time = 0.0, cpu_time = 0.0, gpu_time = 0.0;
      int loop_iterations = 0;
      int i, j, index, loop = 0;
      double delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
      /* TODO: Copy me from transpose version*/
     /*     double **dimObjects = NULL; //calloc_2d(...) -> [numCoords][numObjs]
      double **dimClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
      double **newClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
 */
        double  **dimObjects = (double **)calloc_2d(numCoords, numObjs, sizeof(double));//calloc_2d(...) -> [numCoords][numObjs]
    double  **dimClusters = (double **)calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]
    double  **newClusters = (double **)calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]
      printf("\n|-----------Full-offload GPU Kmeans------------|\n\n");

      /* TODO: Copy me from transpose version*/
     //  for (;;);
      for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j*numCoords + i];
        }
    }
    /* DONE */
      double *deviceObjects;
      double *deviceClusters, *devicenewClusters;
      //look below
      //checkCuda(hipMalloc(&devicenewClusters, numClusters * numCoords * sizeof(double)));
      int *deviceMembership;
      int *devicenewClusterSize; /* [numClusters]: no. objects assigned in each new cluster */

      /* pick first numClusters elements of objects[] as initial cluster centers*/
      for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
          dimClusters[i][j] = dimObjects[i][j];
        }
      }

      /* initialize membership[] */
      for (i = 0; i < numObjs; i++) membership[i] = -1;

      timing = wtime() - timing;
      printf("t_alloc: %lf ms\n\n", 1000 * timing);
      timing = wtime();
      const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize) ? blockSize : numObjs;
      const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock -1) /numThreadsPerClusterBlock ; /* TODO: Calculate Grid size, e.g. number of blocks. */
      /*    Define the shared memory needed per block.
          - BEWARE: We can overrun our shared memory here if there are too many
          clusters or too many coordinates!
          - This can lead to occupancy problems or even inability to run.
          - Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
      const unsigned int clusterBlockSharedDataSize = numCoords *numClusters * sizeof(double);
/*  DONE */
      hipDeviceProp_t deviceProp;
      int deviceNum;
      hipGetDevice(&deviceNum);
      hipGetDeviceProperties(&deviceProp, deviceNum);

      if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
        error("Your CUDA hardware has insufficient block shared memory to hold all cluster centroids\n");
      }

      checkCuda(hipMalloc(&deviceObjects, numObjs * numCoords * sizeof(double)));
      checkCuda(hipMalloc(&deviceClusters, numClusters * numCoords * sizeof(double)));
      checkCuda(hipMalloc(&devicenewClusters, numClusters * numCoords * sizeof(double)));
      checkCuda(hipMalloc(&devicenewClusterSize, numClusters * sizeof(int)));
      checkCuda(hipMalloc(&deviceMembership, numObjs * sizeof(int)));
      checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));

      timing = wtime() - timing;
      printf("t_alloc_gpu: %lf ms\n\n", 1000 * timing);
      timing = wtime();

      checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
                           numObjs * numCoords * sizeof(double), hipMemcpyHostToDevice));
      checkCuda(hipMemcpy(deviceMembership, membership,
                           numObjs * sizeof(int), hipMemcpyHostToDevice));
      checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                           numClusters * numCoords * sizeof(double), hipMemcpyHostToDevice));
      checkCuda(hipMemset(devicenewClusterSize, 0, numClusters * sizeof(int)));
      free(dimObjects[0]);

      timing = wtime() - timing;
      printf("t_get_gpu: %lf ms\n\n", 1000 * timing);
      timing = wtime();

      do {
        timing_internal = wtime();
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));
        timing_gpu = wtime();
        //printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        /* TODO: change invocation if extra parameters needed*/
        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects,  devicenewClusters,devicenewClusterSize, deviceClusters, deviceMembership, dev_delta_ptr);
       /* DONE */

        hipDeviceSynchronize();
        checkLastCudaError();

        gpu_time += wtime() - timing_gpu;

        //printf("Kernels complete for itter %d, updating data in CPU\n", loop);

        timing_transfers = wtime();
        /* TODO: Copy dev_delta_ptr to &delta
          checkCuda(hipMemcpy(...)); */
        checkCuda(hipMemcpy(&delta,dev_delta_ptr, sizeof(double),hipMemcpyDeviceToHost));
        /* DONE */
        transfers_time += wtime() - timing_transfers;

        const unsigned int update_centroids_block_sz = (numCoords * numClusters > blockSize) ? blockSize : numCoords *
                                                                                                           numClusters;  /* TODO: can use different blocksize here if deemed better */
        const unsigned int update_centroids_dim_sz = (numCoords * numClusters + update_centroids_block_sz -1 ) / update_centroids_block_sz; /* TODO: calculate dim for "update_centroids" */
        timing_gpu = wtime();
        /* TODO: use dim for "update_centroids" and fire iti*/
            update_centroids<<< update_centroids_dim_sz, update_centroids_block_sz, 0 >>>
                (numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters);
            /* DONE */
        hipDeviceSynchronize();
        checkLastCudaError();
        gpu_time += wtime() - timing_gpu;

        timing_cpu = wtime();
        delta /= numObjs;
        //printf("delta is %f - ", delta);
        loop++;
        //printf("completed loop %d\n", loop);
        cpu_time += wtime() - timing_cpu;

        timing_internal = wtime() - timing_internal;
        if (timing_internal < timer_min) timer_min = timing_internal;
        if (timing_internal > timer_max) timer_max = timing_internal;
      } while (delta > threshold && loop < loop_threshold);

  checkCuda(hipMemcpy(membership, deviceMembership,
                       numObjs * sizeof(int), hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(dimClusters[0], deviceClusters,
                       numClusters * numCoords * sizeof(double), hipMemcpyDeviceToHost));

  for (i = 0; i < numClusters; i++) {
    for (j = 0; j < numCoords; j++) {
      clusters[i * numCoords + j] = dimClusters[j][i];
    }
  }

  timing = wtime() - timing;
  printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\t"
         "-> t_cpu_avg = %lf ms\n\t-> t_gpu_avg = %lf ms\n\t-> t_transfers_avg = %lf ms\n\n|-------------------------------------------|\n",
         loop, 1000 * timing, 1000 * timing / loop, 1000 * timer_min, 1000 * timer_max,
         1000 * cpu_time / loop, 1000 * gpu_time / loop, 1000 * transfers_time / loop);

  char outfile_name[1024] = {0};
  sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv",
          numObjs * numCoords * sizeof(double) / (1024 * 1024), numCoords, numClusters);
  FILE *fp = fopen(outfile_name, "a+");
  if (!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name);
  fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "All_GPU", blockSize, timing / loop, timer_min, timer_max);
  fclose(fp);

  checkCuda(hipFree(deviceObjects));
  checkCuda(hipFree(deviceClusters));
  checkCuda(hipFree(devicenewClusters));
  checkCuda(hipFree(devicenewClusterSize));
  checkCuda(hipFree(deviceMembership));

  return;
}
